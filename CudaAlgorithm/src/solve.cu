#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <fstream>
#include <iterator>
#include <string>
#include <stack>
#include <algorithm> 
using namespace std;
#include "utils.h"

#define N_THREADS 1024
#define N_BLOCKS 16

/*** GPU functions ***/
/*
 * Intiates random states for each thread
 */
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);
}
/*
 * Attempts to solve the maze and outputs a path
 * If the threads get stuck, they go back to the previous position
 * If the thread gets back to the start with nowhere left to go, then it can't go anywhere else and returns
 */
__global__ void random_solve_path_kernel(char* maze, int rows, int cols, int start, int* path, bool* solved, int* final_pos, hiprandState *state) {
        *solved = false;
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        int curr_pos = start;
        final_pos[tid] = curr_pos;
        float rand_val;
        int old_pos;
        //return;
        //printf("%i", solved);
        path[(curr_pos) + rows * cols * tid] = curr_pos;
        while(*solved != true){

                /**
                 * This part is responsible for generating the legal moves and picking a random one
                 * In an ideal scenario we will have 4 moves, hence why the array is intialized to 4
                 * There are 8 edge cases that are accounted for: [top left corner, top right corner, bottom left corner, bottom right corner, first col but not first row, first row but not first col,
                 * last col but not first row, last row but not first col]
                 */
                 //Keeps track of how many legal_moves we have
            int legal_moves = 0;
                //Intialized to 4 since the max number of legal_moves we can have is 4
            int legal_movesarr[4];
                /**
                 * If the position is in bounds and it isn't the last element in a row (adding a 1 to the last element in a row would mean we zipped to (row + 1, 0), an invalid move)
                 * then it is a legal move to increase curr_pos by 1 (i.e got to the next col) and it is added to the array
                 */

                //printf("%c \n", maze[curr_pos]);
            if(curr_pos + 1 < (rows * cols) && (curr_pos + 1) % cols != 0 && path[(curr_pos + 1) + rows * cols * tid] == -1 && maze[curr_pos + 1] != 'w' ){
                        //printf("We are in loop\n");
                legal_movesarr[legal_moves] = curr_pos + 1;
                legal_moves++;
            }
                /*
                 * If the position is in bounds and it isn't the first element in a row (subtracting a 1 would mean zipping across to the last element in the previous row (row - 1, col - 1), an invalid move),
                 * then it is a legal move to decrease curr_pos by 1 (i.e go to the previous col) and it is added to the array
                 */
            if(curr_pos - 1 >= 0 && curr_pos % cols != 0 && path[(curr_pos - 1) + rows * cols * tid] == -1 && maze[curr_pos -1] != 'w'){
                legal_movesarr[legal_moves] = curr_pos - 1;
                legal_moves++;
            }
                //If it is in bounds, going to the next row (by adding cols) is always legal
            if( curr_pos + cols < rows * cols && path[(curr_pos + cols) + rows * cols * tid] == -1 && maze[curr_pos + cols] != 'w'){
                legal_movesarr[legal_moves] = curr_pos + cols;
                legal_moves++;
            }
                //If it is in bounds, going to the previous row (by subtracting cols) is always legal
            if(curr_pos - cols >= 0 && path[(curr_pos - cols) + rows * cols * tid] == -1 && maze[curr_pos - cols] != 'w'){
                legal_movesarr[legal_moves] = curr_pos - cols;
                legal_moves++;
            }
            //If you backtrack all the way to the start and there is no way to go then you are done
            if(legal_moves == 0 && curr_pos == start){
                return;
            }
            //If you don't have any legal moves, we should go back to where you came from and explore from there
            if(legal_moves == 0){
                curr_pos = path[curr_pos + (rows * cols *tid)];
                continue;
            }

            //Pick random legal move
            rand_val = hiprand_uniform(&state[tid]);
            rand_val *= ((legal_moves - 1) + 0.999999);
            old_pos = curr_pos;
            curr_pos = legal_movesarr[int(rand_val)];
            path[curr_pos + (rows * cols * tid)] = old_pos;
            final_pos[tid] = curr_pos;
	    //Goal found, set solved value to true to signal to all the other the threads that the maze has been solved
            if(maze[curr_pos] == 'g'){
                *solved = true;
            }
        }
}
/*
 * Solves the maze without outputting a path
 * Threads move randomly until they find a goal
 * If the maze is unsolvable, this will run forever
 * It was used as a sanity check to make sure our solving logic was correct
 */
__global__ void random_solve_kernel(char* maze, int rows, int cols, int start, bool* solved, hiprandState *state) {
        *solved = false;
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        int curr_pos = start;
        float rand_val;
        //return;
        //printf("%i", solved);
        while(*solved != true){
            /**
              * This part is responsible for generating the legal moves and picking a random one
              * In an ideal scenario we will have 4 moves, hence why the array is intialized to 4
              * There are 8 edge cases that are accounted for: [top left corner, top right corner, bottom left corner, bottom right corner, first col but not first row, first row but not first col,
              * last col but not first row, last row but not first col]
              */
            //Keeps track of how many legal_moves we have
            int legal_moves = 0;
            //Intialized to 4 since the max number of legal_moves we can have is 4
            int legal_movesarr[4];
            /**
              * If the position is in bounds and it isn't the last element in a row (adding a 1 to the last element in a row would mean we zipped to (row + 1, 0), an invalid move)
              * then it is a legal move to increase curr_pos by 1 (i.e got to the next col) and it is added to the array
              */

             if(curr_pos + 1 < (rows * cols) && (curr_pos + 1) % cols != 0 && maze[curr_pos + 1] != 'w' ){
                 legal_movesarr[legal_moves] = curr_pos + 1;
                 legal_moves++;
             }
             /*
              * If the position is in bounds and it isn't the first element in a row (subtracting a 1 would mean zipping across to the last element in the previous row (row - 1, col - 1), an invalid move),
              * then it is a legal move to decrease curr_pos by 1 (i.e go to the previous col) and it is added to the array
              */
              if(curr_pos - 1 >= 0 && curr_pos % cols != 0 && maze[curr_pos -1] != 'w'){
                  legal_movesarr[legal_moves] = curr_pos - 1;
                  legal_moves++;
               }
               //If it is in bounds, going to the next row (by adding cols) is always legal
               if( curr_pos + cols < rows * cols && maze[curr_pos + cols] != 'w'){
                   legal_movesarr[legal_moves] = curr_pos + cols;
                   legal_moves++;
                }
                //If it is in bounds, going to the previous row (by subtracting cols) is always legal
                if(curr_pos - cols >= 0 && maze[curr_pos - cols] != 'w'){
                    legal_movesarr[legal_moves] = curr_pos - cols;
                    legal_moves++;
                }
                //Pick random legal move
                rand_val = hiprand_uniform(&state[tid]);
                rand_val *= ((legal_moves - 1) + 0.999999);
                curr_pos = legal_movesarr[int(rand_val)];
                //printf("%i \n", curr_pos);

                if(maze[curr_pos] == 'g'){
                        //printf("%i \n", curr_pos);
                        *solved = true;
                }
        }
}

/*** CPU functions ***/

hiprandState* init_rand() {
  hiprandState *d_state;
  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  return d_state;
}

/**
 * Writes out each thread's path unto a line in an output file
 * The output file is used by the visualizer
 */
void writeOut(int* path, int* final_pos, int rows, int cols){
    string finalResult = "";

  //printf("%i\n", ans[0]);
    for(int i = 0; i < N_THREADS * N_BLOCKS; i++){
        vector<int> vec;
        int currIdx = final_pos[i];
        vec.push_back(currIdx);
        while(currIdx != path[currIdx + (rows * cols * i)]){
            vec.push_back(path[currIdx + (rows * cols * i)]);
            currIdx = path[currIdx + (rows * cols * i)];
        }
        reverse(vec.begin(), vec.end());
        stringstream result;
        copy(vec.begin(), vec.end(), ostream_iterator<int>(result, " "));
        string n = result.str();
        finalResult += n += "\n";
    }
    ofstream out("output.txt");
    out << finalResult;

}

/**
 * Classic CPU based DFS solver
 * Used to detetmine whether a maze was solvable or not
 * Used as a sanity check to make sure random_solve_path actually solved the maze
 */
bool dfsSolver(char* maze, int rows, int cols, int start){
   stack<int> myStack;
   //Not actually outputting the path, just using it to make sure we don't revisit nodes we already visited
   //Could use a node struct with a boolean member that determines whether it was visited or not
   //This seemed simpler however
   int* path;
   path = (int *) malloc(sizeof(int) * (rows * cols));
   for(int i = 0; i < (rows * cols); i++){
       path[i] = -1;
   }
   myStack.push(start);
   int curr_pos;
   path[start] = start;
   while(!myStack.empty()){
       //cout << myStack.size() << endl;
       if(maze[myStack.top()] == 'g'){
           return true;
       }
       curr_pos = myStack.top();
       myStack.pop();
       if(curr_pos + cols < rows * cols && path[curr_pos + cols] == -1 && maze[curr_pos + cols] != 'w'){
           myStack.push(curr_pos + cols);
           path[curr_pos + cols] = curr_pos;
       }
       if(curr_pos - cols >= 0 && path[curr_pos - cols] == -1 && maze[curr_pos - cols] != 'w'){
           myStack.push(curr_pos - cols);
           path[curr_pos - cols] = curr_pos;
       }
       if(curr_pos + 1 < rows * cols && path[curr_pos + 1] == -1 && maze[curr_pos + 1] != 'w'){
           myStack.push(curr_pos + 1);
           path[curr_pos + 1] = curr_pos;
       }
       if(curr_pos - 1 >= 0 && path[curr_pos - 1] == -1 && maze[curr_pos - 1] != 'w'){
           myStack.push(curr_pos - 1);
           path[curr_pos - 1] = curr_pos;
       }

   }
   free(path);
   return false;
}
void random_solve_maze(char* maze, int rows, int cols, int start) {
	hiprandState* d_state = init_rand();
	int *path;
	int *d_path;
	int *final_pos;
	int *d_final_pos;
	char *d_maze;
	bool *solved;
	bool *d_solved;
	
	//Allocate memory on CPU
	path = (int *) malloc(sizeof(int) * ((rows * cols) * (N_BLOCKS * N_THREADS)));
	solved = (bool *) malloc(sizeof(bool));
	final_pos = (int *) malloc(sizeof(int) * (N_BLOCKS * N_THREADS));
        //Allocate memory on GPU
	hipMalloc(&d_path, sizeof(int) * ((rows * cols) * (N_BLOCKS * N_THREADS)));
	hipMalloc(&d_maze, sizeof(char) * (rows * cols));
	hipMalloc(&d_solved, sizeof(bool));
	hipMalloc(&d_final_pos, sizeof(int) * (N_BLOCKS * N_THREADS));
	
	for(int i = 0; i < (rows * cols) * (N_BLOCKS * N_THREADS); i++){
		path[i] = -1;
	}

	
	*solved = false;

	//Copy maze over to GPU
	hipMemcpy(d_maze, maze, sizeof(char) * (rows * cols), hipMemcpyHostToDevice);
	hipMemcpy(d_path, path, sizeof(int) * ((rows * cols) * (N_BLOCKS * N_THREADS)), hipMemcpyHostToDevice);
	
        random_solve_path_kernel<<<N_BLOCKS, N_THREADS>>>(d_maze, rows, cols, start, d_path, d_solved, d_final_pos, d_state);

        //Uncomment to use. NOTE: MAZE NEEDS TO BE SOLVABLE
        //random_solve_kernel<<<N_BLOCKS, N_THREADS>>>(d_maze, rows, cols, start, d_solved, d_state);
	
  // After kernel call:
       // Need to copy data back to CPU and check if solved
        hipMemcpy(path, d_path, sizeof(int) * ((rows * cols) * (N_BLOCKS * N_THREADS)), hipMemcpyDeviceToHost);
	hipMemcpy(solved, d_solved, sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(final_pos, d_final_pos, sizeof(int) * N_BLOCKS * N_THREADS, hipMemcpyDeviceToHost);

        //Compares the kernel output with that of the DFS solver
        if(*solved == dfsSolver(maze, rows, cols, start) && *solved == true){
            cout << "Solved\n";
	}
        if(*solved == dfsSolver(maze, rows, cols, start) && *solved == false){
            cout << "Unsolvable\n";
        }
        //If the outputs don't match, then we have a problem was used as a sanity check early on
        if(*solved != dfsSolver(maze, rows, cols, start)){
            cout <<"Error, kernel not working correctly";
            return;
        }
        //Writes out the path
        writeOut(path, final_pos, rows, cols);
        //Frees memeory
	free(solved);
	free(final_pos);
	free(path);
	hipFree(d_final_pos);
	hipFree(d_path);
	hipFree(d_solved);
	hipFree(d_maze);
	hipFree(d_state);

}


int main(int argc, char *argv[]) {
    if (argc != 2) {
       printf("Usage: %s <maze_file> \n", argv[0]);
       return 1;
     }
    //Reads maze file
    ifstream myfile(argv[1]);
    string line;
    getline(myfile, line);
    string space_delimiter = " ";
    vector<string> words;
    
    stringstream ss(line);
    int rows;
    ss >> rows;
    int cols;
    getline(myfile, line);
    stringstream sCols(line);
    sCols >> cols;
    string mazeStr;
    while (getline(myfile, line)){
        mazeStr += line;
    }
    //Gets rid of newline characters and spaces if they are present
    mazeStr.erase(remove(mazeStr.begin(), mazeStr.end(), '\n'),
            mazeStr.end());
    mazeStr.erase(remove(mazeStr.begin(), mazeStr.end(), ' '),
            mazeStr.end());
    char maze[rows * cols];
    strcpy(maze, mazeStr.c_str());
    int start;
    //Finds start position
    for(int i = 0; i < rows * cols; i++){
        if(maze[i] == 's'){
            start = i;
            break;
        }
    }
    random_solve_maze(maze, rows, cols, start);

    return 0;
}
